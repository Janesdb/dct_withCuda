#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>
#include<math.h>
#include<string>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<>
using namespace std;

//#define length 8
#define PI 3.14159265
#define length 8
#define block_len 16

hipError_t dctWithCuda_1(const double *d, double *D);

hipError_t dctWithCuda_2(const double *f, double *F);

/*__global__ void dct1(float *f, float *F){
	int tidy = blockIdx.x*blockDim.x + threadIdx.x;
	int tidx = blockIdx.y*blockDim.y + threadIdx.y;
	int index = tidx*len + tidy;
	float tmp;
	float beta,alfa;
	if(tidx == 0)
		beta = sqrt(1.0/length);
	else
		beta = sqrt(2.0/length);
	if(tidy == 0)
		alfa = sqrt(1.0/length);
	else
		alfa = sqrt(2.0/length);
	if(tidx<length && tidy<length){
		for(i=0; i<length; i++){
			int x = i/length;
			int y = i%length;
			tmp+=((int)data[i])*cos((2*x+1)*tidx*PI/(2.0*length))*
					cos((2*y+1)*tidy*PI/(2.0*length));
		}
		F[index]=(float)alfa*beta*tmp;
	}
}*/

__global__ void dct_1(const double *f,double *F){
	int bid = blockIdx.x;
	//int tid = threadIdx.x;
	int i,j;
	//double data[length]={0.0};
	double tmp;
	if(bid<length){
		__shared__ double data[length];
		for (i=0; i<length; i++)
			data[i] = f[bid*length+i];//load row data from f.
		__syncthreads();
		for(i=0; i<length; i++){
			if(i==0){
				tmp = (double)(1.0/sqrt(1.0*length));
				F[bid] = 0;//why use F[bid]? Do transpose at the same time.
				for(j=0; j<length; j++)
					F[bid] +=data[j] ;
				F[bid] *= tmp;
			}
			else{
				tmp = (double)(sqrt(2.0/(1.0*length)));
				for(i=1; i<length; i++){
					F[i*length+bid] = 0;
					for(j=0; j<length; j++)
						F[i*length+bid] += (double)(data[j]*cos((2*j+1)*i*PI/(2*length)));
					F[i*length+bid] *= tmp;
				}
			}
		}
		__syncthreads();
		for(i=0; i<length; i++)
			data[i] = F[bid*length+1];
		__syncthreads();
		for(i=0; i<length; i++){
			if(i==0){
				tmp=(double)(1.0/sqrt(1.0*length));
				F[bid]=0;
				for(j=0; j<length; j++)
					F[bid] += data[i];
				F[bid] *= tmp;
			}
			else{
				tmp = (double)(sqrt(2.0/(1.0*length)));
				for(i=1; i<length; i++){
					F[i*length+bid] = 0;
					for(j=0; j<length; j++)
						F[i*length+bid] += (double)(data[j]*cos((2*j+1)*i*PI/(2*length)));
					F[i*length+bid] *= tmp;
				}
			}
		}
		__syncthreads();
	}
}

__global__ void dct_2(const double *f, double *F){
	int tidy = blockIdx.x*blockDim.x + threadIdx.x;
	int tidx = blockIdx.y*blockDim.y + threadIdx.y;
	int index = tidx*length + tidy;
	int i;
	double tmp;
	double beta ,alfa;
	if(tidx == 0)
		beta = sqrt(1.0/length);
	else
		beta = sqrt(2.0/length);
	if(tidy == 0)
		alfa = sqrt(1.0/length);
	else
		alfa = sqrt(2.0/length);
	if(tidx<length && tidy<length){
		for(i=0; i<length*length; i++){
			int x = i/length;
			int y = i%length;
			tmp += ((double)f[i])*cos((2*x+1)*tidx*PI/(2.0*length))*
					cos((2*y+1)*tidy*PI/(2.0*length));
		}
		F[index]=(double)alfa * beta * tmp;
	}
}

int main(){
	ifstream infile("/home/zhujian/cuda-workspace/dct_10.16/gradient.txt");
	int i=0;
	string line;
	double f[length*length] = {0.0};
	double F[length*length] = {0.0};
	while(i<length*length){
		if(getline(infile, line)){

			f[i] = atof(line.c_str());
			cout<<"f[i]:  "<<f[i]<<endl;
		}
		i++;
	}
	cout<<"before"<<endl;
	for(i=0; i<length*length; i++){
			cout<<f[i]<<" ";
			if ((i+1)%length==0)
				cout<<endl;
		}
	cout<<endl;
	for(i=0; i<length*length; i++){
			cout<<F[i]<<" ";
			if ((i+1)%length==0)
					cout<<endl;
	}
	hipError_t cudaStatus = dctWithCuda_1(f,F);
	if (cudaStatus != hipSuccess) {
	        fprintf(stderr, "dctWithCuda_1 failed!");
	        return 1;
	}
	cout<<"after"<<endl;
	for(i=0; i<length*length; i++){
		cout<<f[i]<<" ";
		if ((i+1)%length==0)
			cout<<endl;
	}
	cout<<endl;
	for(i=0; i<length*length; i++){
			cout<<F[i]<<" ";
			if ((i+1)%length==0)
					cout<<endl;
	}
	return 0;

}

hipError_t dctWithCuda_1(const double *d, double *D){
	double *dev_d = 0;
	double *dev_D = 0;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if(cudaStatus != hipSuccess){
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_d,length *length* sizeof(double));
	if(cudaStatus != hipSuccess){
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_D,length *length* sizeof(double));
	if(cudaStatus != hipSuccess){
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_d, d,length *length*sizeof(double),hipMemcpyHostToDevice);
	if(cudaStatus != hipSuccess){
		fprintf(stderr, "hipMemcpy-- failed");
		goto Error;
	}
	//launch a kernel on the GPU
	dct_1<<<length,1>>>(dev_d, dev_D);

	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(D, dev_D, length*length* sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }
Error:
	hipFree(dev_d);
	hipFree(dev_D);
	return cudaStatus;
}


hipError_t dctWithCuda_2(const double *d, double *D){
	double *dev_d = 0;
	double *dev_D = 0;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if(cudaStatus != hipSuccess){
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_d,length * sizeof(double));
	if(cudaStatus != hipSuccess){
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_D,length * sizeof(double));
	if(cudaStatus != hipSuccess){
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_d, d,length *sizeof(double),hipMemcpyHostToDevice);
	if(cudaStatus != hipSuccess){
		fprintf(stderr, "hipMalloc failed");
		goto Error;
	}

	//launch a kernel on the GPU
	dct_2<<<1, (length/block_len)*(length/block_len), block_len*block_len>>>(dev_d, dev_D);

	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(D, dev_D, length*length * sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }
Error:
	hipFree(dev_d);
	hipFree(dev_D);

	return cudaStatus;
}
















